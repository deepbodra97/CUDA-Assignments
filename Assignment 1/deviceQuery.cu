#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaCheck.cuh"



int main (int argc, char *argv[])
{

  int numDevices;
  cudaCheck( hipGetDeviceCount(&numDevices));
  std::cout << "Number of devices: " << numDevices << std::endl;

 

  for (int devId = 0; devId < numDevices; devId++){
     std::cout << "-------------------------------------\n";
     hipDeviceProp_t prop;
     cudaCheck( hipGetDeviceProperties(&prop, devId));
     std::cout << "Device id: " << devId << std::endl;
     std::cout << "Device name: " << prop.name << std::endl;
     
/*@  Calculate and print the theoretical peak bandwidth of the
     CUDA device.  You will need to obtain additional information
     from prop.  See the documentation 
*/
     std::cout << "Device memory clock rate: " << prop.memoryClockRate << " KHz" << std::endl;
     std::cout << "Device memory bus width: " << prop.memoryBusWidth << " bits" << std::endl;
     float peakBandwidth = 2.0 * prop.memoryClockRate * prop.memoryBusWidth / 8.0 * 1e-6;
     std::cout << "Device theoretical peak bandwidth: " << peakBandwidth << " GB/s" << std::endl;


  }
  std::cout << "-------------------------------------\n";

}

 
